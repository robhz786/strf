#include "hip/hip_runtime.h"
//  Distributed under the Boost Software License, Version 1.0.
//  (See accompanying file LICENSE_1_0.txt or copy at
//  http://www.boost.org/LICENSE_1_0.txt)

#include "test_utils.hpp"

#include <strf/to_cfile.hpp>

namespace test_utils {

STRF_HD strf::destination<char>*& test_messages_destination_ptr()
{
    static strf::destination<char>* ptr = nullptr;
    return ptr;
}

STRF_HD void set_test_messages_destination(strf::destination<char>& ob)
{
    test_messages_destination_ptr() = &ob;
}

STRF_HD strf::destination<char>& test_messages_destination()
{
    auto * ptr = test_messages_destination_ptr();
    return *ptr;
}

} // namespace test_utils

extern void __device__ run_all_tests();

namespace kernels {

__global__ void kernel_main
    ( unsigned* errors_count
    , char* err_msg
    , std::size_t err_msg_size )
{
    strf::cstr_destination out(err_msg, err_msg_size);
    test_utils::set_test_messages_destination(out);

    run_all_tests ();

    auto result = out.finish();
    (void)result;
    *errors_count = test_utils::test_err_count();
}

} // namespace kernels


int main() {
    auto print = strf::to(stdout);
    int num_devices { 0 };
    hipError_t status = hipGetDeviceCount(&num_devices);

    if (status != hipSuccess) {
        print ("hipGetDeviceCount failed: ", hipGetErrorString(status), '\n');
        return status;
    }
    if (num_devices == 0) {
        print ("No devices - can't run this test\n");
        return status;
    }

    constexpr std::size_t stackSize = 200 * 1024;
    status = hipDeviceSetLimit(hipLimitStackSize, stackSize);
    if (status != hipSuccess) {
        print( "hipDeviceSetLimit(hipLimitStackSize, ", stackSize, ") failed: "
             , hipGetErrorString(status), '\n');
        hipDeviceReset();
        return status;
    }

    constexpr std::size_t buffer_size = 2000;
    struct args {
        unsigned errors_count;
        char buffer[buffer_size];
    };
    struct args* device_side_args;
    status = hipMalloc(&device_side_args, sizeof(struct args));
    if (status != hipSuccess) {
        print("hipMalloc failed: ", hipGetErrorString(status), '\n');
        hipDeviceReset();
        return status;
    }
    status = hipMemset(device_side_args, 0, sizeof(struct args));
    if (status != hipSuccess) {
        print("hipMemset failed: ", hipGetErrorString(status), '\n');
        hipDeviceReset();
        return status;
    }

    int threads_per_block { 1 };
    int blocks_in_grid { 1 };

    kernels::kernel_main<<<threads_per_block, blocks_in_grid>>>(
        &(device_side_args->errors_count),
        &(device_side_args->buffer[0]),
        buffer_size );
    status = hipGetLastError();
    if (status != hipSuccess) {
        print("kernel_main<<<", threads_per_block, ',', blocks_in_grid, ">>> failed: "
             ,hipGetErrorString(status), '\n' );
        return status;
    }
    status = hipDeviceSynchronize();
    // if you get "hipDeviceSynchronize() failed: an illegal memory access was encountered"
    // then try increasing `stackSize` variable above
    if (status != hipSuccess) {
        print("hipDeviceSynchronize() failed: ", hipGetErrorString(status), '\n');
        return status;
    }
    args host_side_args;
    status = hipMemcpy
        ( &host_side_args, device_side_args
        , sizeof(struct args), hipMemcpyDeviceToHost );
    if (status != hipSuccess) {
        print("hipMemcpy failed: ", hipGetErrorString(status), '\n');
        hipDeviceReset();
        return status;
    }
    hipFree(device_side_args);
    hipDeviceReset();

    print (host_side_args.buffer);
    if (host_side_args.errors_count == 0) {
        print("All test passed!\n");
    } else {
        print(host_side_args.errors_count, " tests failed!\n");
    }
    std::fflush(stdout);

    return  host_side_args.errors_count;
}
